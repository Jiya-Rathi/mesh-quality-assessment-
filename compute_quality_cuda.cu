// compute_quality_cuda.cu
// CUDA version: loads a TetGen mesh, computes aspect ratios on the GPU, and records kernel time.

#include <iostream>
#include <fstream>
#include <vector>
#include <array>
#include <cmath>
#include <limits>
#include <hip/hip_runtime.h>

// Simple 3D point
struct Vec3 { double x, y, z; };

// Device function to compute aspect ratio
__device__ double aspectRatioDevice(const Vec3& A, const Vec3& B,
                                    const Vec3& C, const Vec3& D) {
    double minLen = 1e308;
    double maxLen = 0.0;
    // list of edges
    double dx, dy, dz, len;
    // AB
    dx = B.x - A.x; dy = B.y - A.y; dz = B.z - A.z;
    len = sqrt(dx*dx + dy*dy + dz*dz);
    minLen = fmin(minLen, len); maxLen = fmax(maxLen, len);
    // AC
    dx = C.x - A.x; dy = C.y - A.y; dz = C.z - A.z;
    len = sqrt(dx*dx + dy*dy + dz*dz);
    minLen = fmin(minLen, len); maxLen = fmax(maxLen, len);
    // AD
    dx = D.x - A.x; dy = D.y - A.y; dz = D.z - A.z;
    len = sqrt(dx*dx + dy*dy + dz*dz);
    minLen = fmin(minLen, len); maxLen = fmax(maxLen, len);
    // BC
    dx = C.x - B.x; dy = C.y - B.y; dz = C.z - B.z;
    len = sqrt(dx*dx + dy*dy + dz*dz);
    minLen = fmin(minLen, len); maxLen = fmax(maxLen, len);
    // BD
    dx = D.x - B.x; dy = D.y - B.y; dz = D.z - B.z;
    len = sqrt(dx*dx + dy*dy + dz*dz);
    minLen = fmin(minLen, len); maxLen = fmax(maxLen, len);
    // CD
    dx = D.x - C.x; dy = D.y - C.y; dz = D.z - C.z;
    len = sqrt(dx*dx + dy*dy + dz*dz);
    minLen = fmin(minLen, len); maxLen = fmax(maxLen, len);
    return maxLen / minLen;
}

// Kernel: one thread per tetrahedron
__global__ void computeAspectKernel(const Vec3* d_points,
                                    const int4* d_tets,
                                    double* d_ratios,
                                    size_t numTets) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numTets) {
        int4 tet = d_tets[idx];
        Vec3 A = d_points[tet.x];
        Vec3 B = d_points[tet.y];
        Vec3 C = d_points[tet.z];
        Vec3 D = d_points[tet.w];
        d_ratios[idx] = aspectRatioDevice(A, B, C, D);
    }
}

int main() {
    // 1. Load mesh (host)
    std::ifstream nodeFile("diamond1.1.node");
    if (!nodeFile) { std::cerr<<"Cannot open diamond1.1.node\n"; return 1; }
    size_t numPoints; int dim, numAttr, numMarkers;
    nodeFile >> numPoints >> dim >> numAttr >> numMarkers;
    std::vector<Vec3> h_points(numPoints);
    for (size_t i = 0; i < numPoints; ++i) {
        int id; nodeFile >> id >> h_points[i].x >> h_points[i].y >> h_points[i].z;
    }
    nodeFile.close();

    std::ifstream eleFile("diamond1.1.ele");
    if (!eleFile) { std::cerr<<"Cannot open diamond1.1.ele\n"; return 1; }
    size_t numTets; int nodesPerTet, eleMarkers;
    eleFile >> numTets >> nodesPerTet >> eleMarkers;
    std::vector<std::array<int,4>> tets(numTets);
    for (size_t i = 0; i < numTets; ++i) {
        int id, n0, n1, n2, n3;
        eleFile >> id >> n0 >> n1 >> n2 >> n3;
        tets[i] = {{n0-1, n1-1, n2-1, n3-1}};
    }
    eleFile.close();

    // 2. Prepare device data
    Vec3* d_points;
    int4* d_tets;
    double* d_ratios;
    hipMalloc(&d_points, numPoints * sizeof(Vec3));
    hipMalloc(&d_tets,   numTets   * sizeof(int4));
    hipMalloc(&d_ratios, numTets   * sizeof(double));

    hipMemcpy(d_points, h_points.data(), numPoints * sizeof(Vec3), hipMemcpyHostToDevice);
    // convert tets to int4
    std::vector<int4> h_tets_int4(numTets);
    for (size_t i = 0; i < numTets; ++i) {
        auto& tet = tets[i];
        h_tets_int4[i] = make_int4(tet[0], tet[1], tet[2], tet[3]);
    }
    hipMemcpy(d_tets, h_tets_int4.data(), numTets * sizeof(int4), hipMemcpyHostToDevice);

    // 3. Launch kernel & time with CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock = 256;
    int blocks = (numTets + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    computeAspectKernel<<<blocks, threadsPerBlock>>>(d_points, d_tets, d_ratios, numTets);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    double elapsed_s = ms / 1000.0;

    // 4. Copy back and compute host-side stats
    std::vector<double> ratios(numTets);
    hipMemcpy(ratios.data(), d_ratios, numTets * sizeof(double), hipMemcpyDeviceToHost);

    double sum = 0.0;
    double minRatio = std::numeric_limits<double>::infinity();
    double maxRatio = 0.0;
    for (double r : ratios) {
        sum += r;
        minRatio = std::min(minRatio, r);
        maxRatio = std::max(maxRatio, r);
    }
    double avgRatio = sum / numTets;

    // 5. Print results
    std::cout << "GPU: Elapsed_s: " << elapsed_s << "\n";
    std::cout << "Min ratio: " << minRatio
              << ", Avg ratio: " << avgRatio
              << ", Max ratio: " << maxRatio << "\n";

    // 6. Write raw ratios for MATLAB
    std::ofstream fout("aspect_values.txt");
    for (double r : ratios) fout << r << "\n";
    fout.close();

    // 7. Cleanup
    hipFree(d_points);
    hipFree(d_tets);
    hipFree(d_ratios);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
